
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello_GPU(){
    printf("Hello from GPU%d[%d]!\n", blockIdx.x, threadIdx.x);
}

int main(){

    hello_GPU<<<2, 5>>>();
    return 0;
}

